#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define BLOCK_SIZE   512
#define PAGE_SIZE_I  1024   // size of a segment handled by a block (how many integers)
#define PAGE_SIZE_C  (PAGE_SIZE_I*sizeof(int))  // size of a segment handled by a block (how many bytes)
#define EXTRA_SIZE_PER_TB  128   // overlapd region size between segments (unit is integer)
#define CHAR_SET 256

//texture < int, 1, cudaReadModeElementType > tex_r;
//texture < int, 1, cudaReadModeElementType > tex_HT;

// First, look up s_s0Table to jump from initial state to next state.
// If the thread in still alive, then keep tracing HT (hash table)
// in texture until the thread be terminated (-1).
#define  SUBSEG_MATCH( j, match ) \
    pos = tid + j * BLOCK_SIZE ; \
    inputChar = s_in_c[pos]; \
    if (pos < input_size) {\
        state = s_s0Table[inputChar]; \
        yang123 = 0; \
        if (state >= 0) { \
            if (state < num_final_state) { \
                match[yang123] = state; \
                yang123++; \
            } \
            pos += 1; \
            while (1) { \
                if (pos >= bdy) break; \
                inputChar = s_in_c[pos]; \
                int key = (state << 8) + inputChar; \
                int row = key >> width_bit; \
                int col = key & ((1<<width_bit)-1); \
                int index = d_r[row] + col; \
                if(index < 0 || index >= HTSize) \
                    state = -1; \
                else { \
                      int hashValue = d_hash_table[index]; \
                      if ((hashValue) == row) \
                        state = d_val_table[index] ; \
                      else \
                        state = -1; \
                } \
                \
                if (state == -1) break; \
                if (state < num_final_state) { \
                  match[yang123] = state; \
                  yang123++; \
                } \
                if (yang123 > max_pat_len ){ \
                  printf("yang123 is bigger than maxlength in thread%d \n",tid); \
                } \
                pos += 1; \
            } \
        }\
    } 
    

/****************************************************************************
*   Function   : TraceTable_kernel
*   Description: This function trace PHF hash table to match input string
*   Parameters : d_match_result - Address to store match result
*                d_in_i - Device (global) memory in int unit
*                input_size - Size of input string
*                HTSize - Size of hash table
*                width_bit - Bits of key table width
*                num_final_state - Total number of final states
*                MaxRow - Total number of rows in key table
*                num_blocks - Total number of blocks
*                boundary - The last segment size
*                d_s0Table - The row of initial state in PFAC table
*   Returned   : No use
****************************************************************************/
__global__ void TraceTable_kernel(unsigned int *d_match_result, int *d_in_i, int input_size,
                                  int HTSize, int width_bit, int num_final_state, int MaxRow,
                                  int num_blocks, int boundary, int *d_s0Table, int* d_r, int* d_hash_table,
                                  int* d_val_table, int max_pat_len) {
    int tid = threadIdx.x;
    int gbid = blockIdx.y * gridDim.x + blockIdx.x;   // global block ID
    int start = gbid * PAGE_SIZE_I + tid;
    int pos;   // position to read input for the thread
    int state;
    int yang123;
    int inputChar;
    unsigned int *match[(PAGE_SIZE_C / BLOCK_SIZE)] = {0};   // registers to save match result
    for (int i = 0; i < (PAGE_SIZE_C / BLOCK_SIZE); i++) {
        match[i] = (unsigned int*)malloc(sizeof(unsigned int) * max_pat_len);
        for(int j = 0; j < max_pat_len; j++) {
            match[i][j] = - 1;
        }
    }
    unsigned char *s_in_c;   // shared memory in char unit
    unsigned char *d_in_c;   // device (global) memory in char unit
    int bdy;
    __shared__ int s_in_i[PAGE_SIZE_I + EXTRA_SIZE_PER_TB];   // shared memory in int unit
    __shared__ int s_s0Table[CHAR_SET];   // move the row of initial state in PFAC table to shared memory

    if (gbid >= num_blocks) return;

    s_in_c = (unsigned char *) s_in_i;
    d_in_c = (unsigned char *) d_in_i;

    pos = start;
    // move data from global to shared memory
    if(tid < PAGE_SIZE_I + EXTRA_SIZE_PER_TB)
      s_in_i[tid] = d_in_i[pos];
    if(BLOCK_SIZE + tid < PAGE_SIZE_I + EXTRA_SIZE_PER_TB)
      s_in_i[BLOCK_SIZE + tid] = d_in_i[BLOCK_SIZE + pos];
    if (tid < EXTRA_SIZE_PER_TB && 2* BLOCK_SIZE + tid < PAGE_SIZE_I + EXTRA_SIZE_PER_TB) {
        s_in_i[2 * BLOCK_SIZE + tid] = d_in_i[2 * BLOCK_SIZE + pos];
    }
    if (tid < CHAR_SET) {
        s_s0Table[tid] = d_s0Table[tid];
    }
    __syncthreads();

    if (gbid == num_blocks - 1)
        bdy = boundary;
    else
        bdy = PAGE_SIZE_C + EXTRA_SIZE_PER_TB * sizeof(int);

    //  every thread handle (PAGE_SIZE_C/BLOCK_SIZE = 8) position
    SUBSEG_MATCH(0, match[0]);
    SUBSEG_MATCH(1, match[1]);
    SUBSEG_MATCH(2, match[2]);
    SUBSEG_MATCH(3, match[3]);
    SUBSEG_MATCH(4, match[4]);
    SUBSEG_MATCH(5, match[5]);
    SUBSEG_MATCH(6, match[6]);
    SUBSEG_MATCH(7, match[7]);

    // save match result from registers to global memory
    start = gbid * PAGE_SIZE_C + tid;
    unsigned int d_match_size = (unsigned int)max_pat_len * (unsigned int)input_size;
    unsigned int thread_offset = (unsigned int)start * (unsigned int)max_pat_len; 
    #pragma unroll
    for (int i = 0; i < 8; i++) {
        unsigned int i_offset = (unsigned int)i * (unsigned int)max_pat_len * (unsigned int)BLOCK_SIZE;
        for (int j = 0; j < max_pat_len; j++) {
            if(thread_offset + i_offset + (unsigned int)j < 0) printf("Overflow??\n");
            if(thread_offset + i_offset + (unsigned int)j < d_match_size) {
              d_match_result[thread_offset + i_offset + (unsigned int)j] = match[i][j];
            }
            if(int(match[i][j])<-1) printf("???\n");
        }
        free(match[i]);
    }
}

// First, look up s_s0Table to jump from initial state to next state.
// If the thread in still alive, then keep tracing HT (hash table)
// in texture until the thread be terminated (-1).
#define  SUBSEG_MATCH_FAST( j, match ) \
    pos = tid + j * BLOCK_SIZE ; \
    inputChar = s_in_c[pos]; \
    state = s_s0Table[inputChar]; \
    yang123 = 0; \
    if (state >= 0) { \
        if (state < num_final_state) { \
            match[yang123] = state; \
            yang123++; \
        } \
        pos += 1; \
        while (1) { \
            if (pos >= bdy) break; \
            inputChar = s_in_c[pos]; \
            int index = d_r[state] + inputChar; \
            if (index >= HTSize) \
                state = -1; \
            else { \
                int hashValue = d_hash_table[index]; \
                if (hashValue == state) \
                    state = d_val_table[index] ; \
                else \
                    state = -1; \
            } \
            \
            if (state == -1) break; \
            if (state < num_final_state) { \
            match[yang123] = state; \
            yang123++; \
            } \
            pos += 1; \
        } \
    }

///****************************************************************************
//*   Function   : TraceTable_kernel_fast
//*   Description: This function trace PHF hash table to match input string.
//*                Because the width of key table is 256, some computation
//*                can be discarded.
//*   Parameters : d_match_result - Address to store match result
//*                d_in_i - Device (global) memory in int unit
//*                input_size - Size of input string
//*                HTSize - Size of hash table
//*                width_bit - Bits of key table width
//*                num_final_state - Total number of final states
//*                MaxRow - Total number of rows in key table
//*                num_blocks - Total number of blocks
//*                boundary - The last segment size
//*                d_s0Table - The row of initial state in PFAC table
//*   Returned   : No use
//****************************************************************************/
//__global__ void TraceTable_kernel_fast(unsigned int *d_match_result, int *d_in_i,
//                                       int input_size, int HTSize, int num_final_state, int MaxRow,
//                                       int num_blocks, int boundary, int *d_s0Table,
//                                       int* d_r, int* d_hash_table, int* d_val_table, int max_pat_len) {
//    int tid = threadIdx.x;
//    int gbid = blockIdx.y * gridDim.x + blockIdx.x;   // global block ID
//    int start = gbid * PAGE_SIZE_I + tid;
//    int pos;   // position to read input for the thread
//    int state;
//    int yang123;
//    int inputChar;
//    unsigned int match[(PAGE_SIZE_C / BLOCK_SIZE)][100] = {0};   // registers to save match result
//    unsigned char *s_in_c;   // shared memory in char unit
//    unsigned char *d_in_c;   // device (global) memory in char unit
//    int bdy;
//    __shared__ int s_in_i[PAGE_SIZE_I + EXTRA_SIZE_PER_TB];   // shared memory in int unit
//    __shared__ int s_s0Table[CHAR_SET];   // move the row of initial state in PFAC table to shared memory
//
//    if (gbid >= num_blocks) return;
//
//    s_in_c = (unsigned char *) s_in_i;
//    d_in_c = (unsigned char *) d_in_i;
//
//    pos = start;
//    // move data from global to shared memory
//    s_in_i[tid] = d_in_i[pos];
//    s_in_i[BLOCK_SIZE + tid] = d_in_i[BLOCK_SIZE + pos];
//    if (tid < EXTRA_SIZE_PER_TB) {
//        s_in_i[2 * BLOCK_SIZE + tid] = d_in_i[2 * BLOCK_SIZE + pos];
//    }
//    if (tid < CHAR_SET) {
//        s_s0Table[tid] = d_s0Table[tid];
//    }
//    __syncthreads();
//
//    if (gbid == num_blocks - 1)
//        bdy = boundary;
//    else
//        bdy = PAGE_SIZE_C + EXTRA_SIZE_PER_TB * sizeof(int);
//
//    // every thread handle (PAGE_SIZE_C/BLOCK_SIZE) position
//    SUBSEG_MATCH_FAST(0, match[0]);
//    SUBSEG_MATCH_FAST(1, match[1]);
//    SUBSEG_MATCH_FAST(2, match[2]);
//    SUBSEG_MATCH_FAST(3, match[3]);
//    SUBSEG_MATCH_FAST(4, match[4]);
//    SUBSEG_MATCH_FAST(5, match[5]);
//    SUBSEG_MATCH_FAST(6, match[6]);
//    SUBSEG_MATCH_FAST(7, match[7]);
//
//    // save match result from registers to global memory
//    start = gbid * PAGE_SIZE_C + tid;
//#pragma unroll
//    for (int i = 0; i < 8; i++) {
//        for (int j = 0; j < max_pat_len; j++) {
//            d_match_result[start*max_pat_len + i*max_pat_len*BLOCK_SIZE + j] = match[i][j];
//        }
//    }
//}

/****************************************************************************
*   Function   : GPU_TraceTable
*   Description: This function prepapre resources for GPU, and launch kernel
*                according to the width of key table
*   Parameters : input_string - Input string
*                input_size - Size of input string
*                state_num - Total number of statesPFAC_table.txt
*                final_state_num - Total number of final states
*                match_result - Address to store match result
*                HTSize - Size of hash table
*                width - The width of key table
*                s0Table - The row of initial state in PFAC table
*   Returned   : No use
****************************************************************************/
//int GPU_TraceTable(unsigned char *input_string, int input_size, int state_num,
//                   int final_state_num, unsigned int* match_result, int HTSize, int width,
//                   int *s0Table, int max_pat_len, int r[], int HT[], int val[])
//{
//
//
//    cudaGetDeviceCount(&GPU_S);
//
//
//
//
//
//
//
//        printf("1. H2D transfer time: %lf ms\n", transInTime);
//        printf("   H2D throughput: %lf GBps\n", (input_size+MaxRow*sizeof(int)+HTSize*sizeof(int)+CHAR_SET*sizeof(int))
//                                                /(transInTime*1000000));
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after malloc memory7: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//
//
//
//         //size_t free_mem, total_mem ;
//         //cudaError_t mem_info = cudaMemGetInfo( &free_mem, &total_mem);
//         //if ( cudaSuccess != mem_info ) {
//         //    printf("memory get info fails\n");
//         //    exit(1) ;
//         //}
//         //printf("total mem = %lf MB, free mem = %lf MB \n", total_mem/1024.0/1024.0 , free_mem/1024.0/1024.0 );
//
//        // set texture memory for hash table on device
//        // cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc <int> ();  // another usage
//
//
//
//
//
//
//
////
////        cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc (sizeof(int)*8, 0, 0, 0, cudaChannelFormatKindSigned);
////        cuda_err = cudaBindTexture(0, tex_r, d_r, channelDesc, MaxRow*sizeof(int));
////        if ( cudaSuccess != cuda_err ){
////            printf("cudaBindTexture on tex_r error\n");
////            exit(1) ;
////        }
////
////        cuda_err = cudaBindTexture(0, tex_HT, d_hash_table, channelDesc, HTSize*sizeof(int));
////        if ( cudaSuccess != cuda_err ){
////            printf("cudaBindTexture on tex_HT error\n");
////            exit(1) ;
////        }
//
//        // count bit of width (ex: if width is 256, width_bit is 8)
//
//
//
//        // release memory
//        cudaFree(d_input_string);
//        printf("cudaFree(d_input_string); done\n");
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory1: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        //cudaUnbindTexture(tex_r);
//        cudaFree(d_r);
//        printf("cudaFree(d_r); done\n");
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory2: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        //cudaUnbindTexture(tex_HT);
//        cudaFree(d_hash_table);
//        printf("cudaFree(d_hash_table); done\n");
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory3: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        cudaFree(d_val_table);//add by qiao0324
//        printf("cudaFree(d_val_table); done\n");
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory4: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        cudaFree(d_match_result);
//        printf("cudaFree(d_match_result); done\n");
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory5: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        cudaFree(d_s0Table);
//        printf("cudaFree(d_s0Table); done\n");
//
//
//        cuda_err = cudaGetLastError() ;
//        if ( cudaSuccess != cuda_err ) {
//            printf("after free memory6: error = %s\n", cudaGetErrorString (cuda_err));
//            exit(1) ;
//        }
//        // for(int testindex = 0; testindex < sizeof(short)*max_pat_len*input_size; testindex ++) {
//        //   if(match_result[testindex] < -1) printf("2Negative value %d at index %d\n", match_result[testindex], testindex);
//        // }
//
//        return 0 ;
//
//
//}
